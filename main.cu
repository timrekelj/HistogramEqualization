#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "helpers/stb_image.h"
#include "helpers/stb_image_write.h"

#define COLOR_CHANNELS 1

#define GRAYLEVELS 256
#define DESIRED_NCHANNELS 1

__global__ void histogram_kernel(unsigned char* image, int width, int height, unsigned int* histogram) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int lx = threadIdx.x;
    int ly = threadIdx.y;

    __shared__ unsigned int localHistogram[GRAYLEVELS];

    localHistogram[blockDim.x * ly + lx] = 0;
    __syncthreads();

    if (x < width && y < height) {
        atomicAdd(&localHistogram[image[y * width + x]], 1);
    }

    __syncthreads();

    atomicAdd(&(histogram[blockDim.x * ly + lx]), localHistogram[blockDim.x * ly + lx]);
}

__global__ void CDF_kernel(unsigned int* histogram, unsigned int* cdf) {
    __shared__ unsigned int temp[GRAYLEVELS * 2];
    int tid = threadIdx.x;

    int pout = 0, pin = 1;

    temp[tid] = histogram[tid];

    __syncthreads();

    for (int offset = 1; offset < GRAYLEVELS; offset <<= 1) {
        pout = 1 - pout;
        pin = 1 - pout;
        if (tid >= offset)
            temp[pout*GRAYLEVELS+tid] = temp[pin*GRAYLEVELS+tid] + temp[pin*GRAYLEVELS+tid - offset];
        else
            temp[pout*GRAYLEVELS+tid] = temp[pin*GRAYLEVELS+tid];
        __syncthreads();
    }

    cdf[tid] = temp[pout*GRAYLEVELS+tid];
}

__device__ unsigned int findMin(unsigned int* cdf){
    unsigned int min = 0;
    for (int i = 0; min == 0 && i < GRAYLEVELS; i++) {
		min = cdf[i];
    }
    
    return min;
}

__device__ unsigned char Scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize){
    float scale;
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    scale = round(scale * (float)(GRAYLEVELS-1));
    return (int)scale;
}

__global__ void equalize_kernel(unsigned char* image_in, unsigned char* image_out, int width, int height, unsigned int* cdf) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
        image_out[y * width + x] = Scale(cdf[image_in[y * width + x]], findMin(cdf), width * height);
}

float run(unsigned char* imageIn, int width, int height, int cpp) {
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    //Allocate memory for raw output image data, histogram, and CDF 
	unsigned char *imageOut = (unsigned char *)malloc(height * width * sizeof(unsigned int));
    unsigned int *histogram = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
    unsigned int *CDF = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));

    unsigned char *imageIn_cuda;
    unsigned char *imageOut_cuda;
    unsigned int *histogram_cuda;
    unsigned int *CDF_cuda;
    hipMalloc((void **)&imageIn_cuda, width * height * sizeof(unsigned char));
    hipMalloc((void **)&imageOut_cuda, width * height * sizeof(unsigned char));
    hipMalloc((void **)&histogram_cuda, GRAYLEVELS * sizeof(unsigned int));
    hipMalloc((void **)&CDF_cuda, GRAYLEVELS * sizeof(unsigned int));

    hipMemcpy(imageIn_cuda, imageIn, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // 1. Izračun histograma:
    dim3 blockDim(16,16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    histogram_kernel<<<gridDim, blockDim>>>(imageIn_cuda, width, height, histogram_cuda);
    hipDeviceSynchronize();
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA histogram error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    CDF_kernel<<<1, GRAYLEVELS, GRAYLEVELS * sizeof(unsigned int)>>>(histogram_cuda, CDF_cuda);
    hipDeviceSynchronize();
    cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA CDF error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    equalize_kernel<<<gridDim, blockDim>>>(imageIn_cuda, imageOut_cuda, width, height, CDF_cuda);
    hipDeviceSynchronize();
    cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA equalize error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    hipMemcpy(imageOut, imageOut_cuda, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

    stbi_write_jpg("images-output/output.jpg", width, height, DESIRED_NCHANNELS, imageOut, 100);

    free(imageOut);
    free(histogram);
    free(CDF);
    hipFree(imageIn_cuda);
    hipFree(imageOut_cuda);
    hipFree(histogram_cuda);
    hipFree(CDF_cuda);

    return elapsedTime;
}

int main(int argc, char** argv){
    // Read image from file
    int width, height, cpp;
    // read only DESIRED_NCHANNELS channels from the input image:
    unsigned char *imageIn = stbi_load(argv[1], &width, &height, &cpp, DESIRED_NCHANNELS);
    if(imageIn == NULL) {
        printf("Error in loading the image\n");
        return 1;
    }

    float timeSum = 0;
    const int REPETITIONS = atoi(argv[2]);
    for (int i = 0; i < REPETITIONS; i++)
    {
        timeSum += run(imageIn, width, height, cpp);
    }
    printf("Average time: %.2f ms\n", timeSum/REPETITIONS);

    free(imageIn);
}
